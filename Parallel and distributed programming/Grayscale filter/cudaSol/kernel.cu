#include "hip/hip_runtime.h"
#include "..\Common\Image.h"
#include "..\Common\Filter.h"

#include "hip/hip_runtime.h"
#include ""

#include "cudaUtils.cuh"

#include <chrono>
#include <exception>
#include <functional>
#include <iostream>
#include <memory>

#define RED     0
#define GREEN   1
#define BLUE    2
#define OPACITY 3

const size_t Blocks = 2048;
const size_t ThreadsPerBlock = 512;

__device__ uint8_t interval(int32_t value)
{
	if (value > 255) return 255;
	if (value < 0) return 0;
	return (uint8_t)value;
}

__global__ void applyFilter(
	uint8_t* image, size_t height, size_t width,
	size_t borderedHeight, size_t borderedWidth,
	int32_t* filter, size_t filterSize, uint8_t* result
)
{
	uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	uint32_t index = x + y * blockDim.x * gridDim.x;
	//uint32_t blockId = blockIdx.x + blockIdx.y * gridDim.x;

	while (index < height * width)
	{
		int32_t red = 0, green = 0, blue = 0;
		size_t h = index / width;
		size_t w = index % width;

		index += blockDim.x * blockDim.y * gridDim.x * gridDim.y;

		for (size_t i = 0; i < filterSize; i++)
			for (size_t j = 0; j < filterSize; j++)
			{
				uint8_t* pixel = image + 4 * (h + i) * borderedWidth + 4 * (w + j);
				red += pixel[RED] * filter[i * filterSize + j];
				green += pixel[GREEN] * filter[i * filterSize + j];
				blue += pixel[BLUE] * filter[i * filterSize + j];
			}

		uint8_t* pixel = result + 4 * h * width + 4 * w;
		pixel[RED] = interval(red);
		pixel[GREEN] = interval(green);
		pixel[BLUE] = interval(blue);
		pixel[OPACITY] = 255;
	}

}

int main()
{
	batman::Image image;
	batman::Image result;

	std::string initialFile = "Polar_Bear_Svalbard_Norwegian_Arctic.png";
	std::string resultFile = "Result.png";

	try
	{
		image.LoadFromFile(initialFile);
		std::cout << "Loaded image " << initialFile << std::endl;

		image.Border(border);

		result.Resize(image.Height(), image.Width());

		size_t gpuFilterSize = filterSize;
		size_t gpuImageHeight = image.Height();
		size_t gpuImageWidth = image.Width();
		size_t gpuImageBorderedHeight = image.BorderedHeight();
		size_t gpuImageBorderedWidth = image.BorderedWidth();

		auto gpuImageBuffer = std::unique_ptr<uint8_t, std::function<void(uint8_t*)>>(
			reinterpret_cast<uint8_t*>(batman::cudaAlloc(image.BufferSize())),
			[](uint8_t* ptr) { hipFree(ptr); }
		);

		auto gpuFilterBuffer = std::unique_ptr<int32_t, std::function<void(int32_t*)>>(
			reinterpret_cast<int32_t*>(batman::cudaAlloc(filterSize * filterSize * sizeof(int32_t))),
			[](int32_t* ptr) { hipFree(ptr); }
		);

		auto gpuResultBuffer = std::unique_ptr<uint8_t, std::function<void(uint8_t*)>>(
			reinterpret_cast<uint8_t*>(batman::cudaAlloc(result.BufferSize())),
			[](uint8_t* ptr) { hipFree(ptr); }
		);

		hipError_t error = hipMemcpy(gpuImageBuffer.get(), image.Buffer(), image.BufferSize(), hipMemcpyKind::hipMemcpyHostToDevice);
		if (error != hipError_t::hipSuccess)
		{
			throw new std::runtime_error("hipMemcpy failed with error code" + std::to_string(error));
		}

		for (int i = 0; i < filterSize; i++)
		{
			error = hipMemcpy(
				gpuFilterBuffer.get() + i * filterSize,
				filter[i],
				filterSize * sizeof(int32_t),
				hipMemcpyKind::hipMemcpyHostToDevice
			);
			if (error != hipError_t::hipSuccess)
			{
				throw new std::runtime_error("hipMemcpy failed with error code" + std::to_string(error));
			}
		}

		__int64 start = std::chrono::duration_cast<std::chrono::milliseconds>(
			std::chrono::system_clock::now().time_since_epoch()
			).count();

		applyFilter << <Blocks, ThreadsPerBlock >> > (
			gpuImageBuffer.get(), gpuImageHeight, gpuImageWidth,
			gpuImageBorderedHeight, gpuImageBorderedWidth,
			gpuFilterBuffer.get(), gpuFilterSize, gpuResultBuffer.get()
			);

		error = hipMemcpy(result.Buffer(), gpuResultBuffer.get(), result.BufferSize(), hipMemcpyKind::hipMemcpyDeviceToHost);
		if (error != hipError_t::hipSuccess)
		{
			throw new std::runtime_error("hipMemcpy failed with error code" + std::to_string(error));
		}

		__int64 stop = std::chrono::duration_cast<std::chrono::milliseconds>(
			std::chrono::system_clock::now().time_since_epoch()
			).count();

		std::cout << "Finished aplying filter on " << Blocks << " blocks and "
			<< ThreadsPerBlock << " threads per blocks in "
			<< stop - start << " miliseconds" << std::endl;

		image.Unborder();

		//image.StoreToFile("Initial.png");
		result.StoreToFile(resultFile);
		std::cout << "Stored result in image " << resultFile << std::endl;
	}
	catch (std::exception& ex)
	{
		std::cerr << ex.what() << std::endl;
		return -1;
	}
}
